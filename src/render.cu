#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/random.h>
#include "render.h"
#include "tools.h"

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

#define BACKFACECULLING 1
#define DEBUG_STATISTICS 0 

#include "helpers.h"
#include "pipeline/vertexshader.cu"
#include "pipeline/primitiveassembly.cu"
#include "pipeline/rasterizer.cu"
#include "pipeline/fragmentshader.cu"

//Writes fragment colors to the framebuffer
__global__ void render(glm::vec2 resolution, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if(x<=resolution.x && y<=resolution.y){
    framebuffer[index] = depthbuffer[index].color;
  }
}

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRasterizeCore(glm::mat4 view, glm::mat4 projection, glm::vec3 light, int draw_mode, uchar4* PBOpos, glm::vec2 resolution, float frame, float* vbo, int vbosize, float* nbo, int nbosize, float* cbo, int cbosize, int* ibo, int ibosize){

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(resolution.x)/float(tileSize)), (int)ceil(float(resolution.y)/float(tileSize)));

  //set up framebuffer
  framebuffer = NULL;
  hipMalloc((void**)&framebuffer, (int)resolution.x*(int)resolution.y*sizeof(glm::vec3));

  //set up depthbuffer
  depthbuffer = NULL;
  hipMalloc((void**)&depthbuffer, (int)resolution.x*(int)resolution.y*sizeof(fragment));

  depth = NULL;
  hipMalloc((void**)&depth, (int)resolution.x*(int)resolution.y*sizeof(unsigned int));

  //kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
  clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, framebuffer, glm::vec3(0,0,0));

  fragment frag;
  frag.color = glm::vec3(0,0,0);
  frag.normal = glm::vec3(0,0,0);
  frag.position = glm::vec3(0,0,-10000);
  frag.lock = 0;
  frag.z = -FLT_MAX;
  clearDepthBuffer<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer, depth, frag);

  //------------------------------
  //memory stuff
  //------------------------------
  primitives = NULL;
  hipMalloc((void**)&primitives, (ibosize/3)*sizeof(triangle));

  vertices = NULL;
  hipMalloc((void**)&vertices, (ibosize)*sizeof(vertex));

  device_ibo = NULL;
  hipMalloc((void**)&device_ibo, ibosize*sizeof(int));
  hipMemcpy( device_ibo, ibo, ibosize*sizeof(int), hipMemcpyHostToDevice);

  device_vbo = NULL;
  hipMalloc((void**)&device_vbo, vbosize*sizeof(float));
  hipMemcpy( device_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice);

  device_vbo_eye = NULL;
  hipMalloc((void**)&device_vbo_eye, vbosize*sizeof(float));

  device_nbo = NULL;
  hipMalloc((void**)&device_nbo, nbosize*sizeof(float));
  hipMemcpy( device_nbo, nbo, nbosize*sizeof(float), hipMemcpyHostToDevice);

  device_cbo = NULL;
  hipMalloc((void**)&device_cbo, cbosize*sizeof(float));
  hipMemcpy( device_cbo, cbo, cbosize*sizeof(float), hipMemcpyHostToDevice);

#if DEBUG_STATISTICS
  numCulledTriangles = NULL;
  hipMalloc((void**)&numCulledTriangles, sizeof(int));
#endif

  tileSize = 32;
  int primitiveBlocks = ceil(((float)vbosize/3)/((float)tileSize));

  //------------------------------
  //vertex shader
  //------------------------------
  vertexShadeKernel<<<primitiveBlocks, tileSize>>>(view, projection, light, device_vbo, vbosize, device_nbo, nbosize, vertices);

  hipDeviceSynchronize();
  //------------------------------
  //primitive assembly
  //------------------------------
  primitiveBlocks = ceil(((float)ibosize/3)/((float)tileSize));
  primitiveAssemblyKernel<<<primitiveBlocks, tileSize>>>(vertices, device_cbo, cbosize, device_ibo, ibosize, primitives);
  hipDeviceSynchronize();
  //------------------------------
  //rasterization
  //------------------------------
  rasterizationKernel<<<primitiveBlocks, tileSize>>>(primitives, ibosize/3, depthbuffer, depth, resolution, numCulledTriangles);
#if DEBUG_STATISTICS
  int * host_CulledT;
  hipHostAlloc((void**) &host_CulledT, sizeof(int), hipHostMallocDefault);
  hipMemcpy( host_CulledT, numCulledTriangles, sizeof(int), hipMemcpyDeviceToHost);
  printf("Number of Culled Triangles: %d", *host_CulledT);
#endif
  hipDeviceSynchronize();
  //------------------------------
  //fragment shader
  //------------------------------
  fragmentShadeKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(depthbuffer, resolution, draw_mode);
  hipDeviceSynchronize();
  //------------------------------
  //write fragments to framebuffer
  //------------------------------
  render<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer, framebuffer);
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, resolution, framebuffer);

  hipDeviceSynchronize();

  kernelCleanup();

  checkCUDAError("Kernel failed!");
}

void kernelCleanup(){
  hipFree( primitives );
  hipFree( vertices );
  hipFree( device_vbo );
  hipFree( device_cbo );
  hipFree( device_ibo );
  hipFree( framebuffer );
  hipFree( depthbuffer );
  hipFree( device_nbo );
  hipFree( device_vbo_eye );
  hipFree( depth );
}
